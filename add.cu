#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c){
	*c = *a + *b;
}


int main(void){
	int ha=1, hb=2, hc;
	//add<<<1,1>>>(&ha, &hb, &hc);
	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, sizeof(int));
	hipMalloc((void **)&d_b, sizeof(int));
	hipMalloc((void **)&d_c, sizeof(int));

	hipMemcpy(d_a, &ha, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &hb, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, &hc, sizeof(int), hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&hc, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("hc = %d\n", hc);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("hc = %d\n", hc);


	return 0;
}
