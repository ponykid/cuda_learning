#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello( void ){
	printf("hello, cuda\n");
}

__global__ void helloCuda( float *a ){
	printf("hello ! data = %f \n", *a);
}

int main( void ){
	printf("hello world from CPU \n");

	// kernel_function<<<num_blocks, num_thread>>>(param1,param2)
	hello<<<1,10>>>();

	hipDeviceReset();

	float h_a = 1;
	float *d_a;
	hipMalloc(&d_a, sizeof(float));
	hipMemcpy(d_a, &h_a, sizeof(float), hipMemcpyHostToDevice);

	helloCuda<<<1,10>>>(d_a);
	hipMemcpy(&h_a, d_a, sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
	//cudaDeviceSynchronize();

	hipFree(&d_a);

	int size = 1<<24;

	printf("size = %d\n", size);

	return 0;
}
